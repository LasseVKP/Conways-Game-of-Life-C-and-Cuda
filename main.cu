#include "hip/hip_runtime.h"
// Use raylib and raygui for graphics
#include "raylib.h"
#define RAYGUI_IMPLEMENTATION
#include "raygui.h"
#include <hiprand/hiprand_kernel.h>

// Set window and board sizes
#define BOARD_SIZE 100
#define WINDOW_SIZE 1000
#define CELL_SIZE (WINDOW_SIZE/BOARD_SIZE)

// One thread per cell
#define THREADS BOARD_SIZE*BOARD_SIZE
#define THREADS_PER_BLOCK 256
#define BLOCKS (THREADS + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK

// Population and generation label properties
#define TEXT_SIZE 48
#define TEXT_COLOR DARKBLUE

int population = 0;
int generation = 0;

// Give a randomness state to every thread
__global__ void initCurand(hiprandState *states, unsigned long seed){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure no threads outside cell total do anything
    if(idx>=BOARD_SIZE*BOARD_SIZE) return;

    hiprand_init(seed, idx, 0, &states[idx]);
}

// Randomize all cells and count population
__global__ void initBoard(bool *board, hiprandState *states) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Make sure no threads outside cell total do anything
    if(idx>=BOARD_SIZE*BOARD_SIZE) return;

    // Generate a random number between 0-1 based on thread state
    float state = hiprand_uniform(&states[idx]);
    // Convert to a bool
    board[idx] = (state > 0.5f);
}

__global__ void updateBoard(bool *board){

}

// Draw all the cells
void drawBoard(bool *board){
    for (size_t i = 0; i < BOARD_SIZE*BOARD_SIZE; i++)
    {
        if(board[i]){
            DrawRectangle((i%BOARD_SIZE)*CELL_SIZE, i/BOARD_SIZE*CELL_SIZE, CELL_SIZE, CELL_SIZE, BLACK);
        }
    }
}

// Draw population count and generation count
void drawLabel(){
    char populationText[24];
    sprintf(populationText, "Population: %d", population);
    DrawText(populationText, 5, 5, TEXT_SIZE, TEXT_COLOR);

    char generationText[24];
    sprintf(generationText, "Generation: %d", generation);
    DrawText(generationText, 5, 5 + TEXT_SIZE, TEXT_SIZE, TEXT_COLOR);
}

int main(void) {
    // Open the window
    InitWindow(WINDOW_SIZE, WINDOW_SIZE, "Conway's Game Of Life");
    SetTargetFPS(2);

    // Setup randomness on the device
    hiprandState *d_states;
    hipError_t err = hipMalloc((void**)&d_states, sizeof(hiprandState)*THREADS);
    if (err != hipSuccess) printf("hipMalloc states failed: %s\n", hipGetErrorString(err));
    initCurand<<<BLOCKS, THREADS_PER_BLOCK>>>(d_states, time(NULL));

    // Setup the board on the device
    bool *d_board;

    // Allocate the board to the device memory
    hipMalloc((void**)&d_board, sizeof(bool)*BOARD_SIZE*BOARD_SIZE);

    initBoard<<<BLOCKS, THREADS_PER_BLOCK>>>(d_board, d_states);

    // Transfer board to host memory
    bool *h_board = (bool*)malloc(sizeof(bool) * BOARD_SIZE * BOARD_SIZE);
    hipMemcpy(h_board, d_board, sizeof(bool) * BOARD_SIZE * BOARD_SIZE, hipMemcpyDeviceToHost);
    drawBoard(h_board);

    // Main loop
    while (!WindowShouldClose()) {
        // Update and draw
        BeginDrawing();
        ClearBackground(WHITE);
        initBoard<<<BLOCKS,THREADS_PER_BLOCK>>>(d_board, d_states);
        hipDeviceSynchronize();
        hipMemcpy(h_board, d_board, sizeof(bool) * BOARD_SIZE * BOARD_SIZE, hipMemcpyDeviceToHost);

        drawBoard(h_board);
        drawLabel();
        EndDrawing();
        // Increment generation counter and reset population counter
        generation++;
        population = 0;
    }

    // Clear allocated memory
    free(h_board);
    hipFree(d_board);
    hipFree(d_states);

    CloseWindow();
}
